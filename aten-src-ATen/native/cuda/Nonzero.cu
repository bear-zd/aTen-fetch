#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/EmptyTensor.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/EmptyTensor.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <ATen/cuda/cub.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh> //for MAX_DIMS

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty_native.h>
#include <ATen/ops/nonzero_native.h>
#endif

namespace at::native {

namespace {
template <typename T>
struct NonZeroOp {
  __host__ __device__ __forceinline__ bool operator()(const T& a) const {
    return (a != T(0));
  }
};

// TODO: actually support int64_t index_t
template <typename index_t>
struct TensorDims {
  index_t sizes[MAX_DIMS];
};

template <typename index_t>
__global__ void write_indices(
    int64_t* inp,
    TensorDims<index_t> dims,
    int ndim,
    index_t n) {
  auto index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    index_t div = 1;
    int64_t idx_flat = inp[index];
#pragma unroll
    for (int dim = MAX_DIMS; dim >= 0; dim--) {
      if (dim > ndim - 1)
        continue;
      auto dim_size = dims.sizes[dim];
      inp[index + dim * n] = (idx_flat / div) % dim_size;
      div *= dim_size;
    }
  }
}

} // anonymous namespace

template <typename scalar_t>
void nonzero_cuda_out_impl(const Tensor& self, Tensor& out) {
  Tensor self_ = self.contiguous();
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  int64_t chunk_size, num_chunks;
  if (self.numel() < std::numeric_limits<int>::max()) {
    chunk_size = self.numel();
    num_chunks = 1;
  } else {
    chunk_size = std::numeric_limits<int>::max() / 2 + 1; // 2**30
    num_chunks = (self.numel() + chunk_size - 1) / chunk_size;
  }
  // compute number of nonzero elements
  size_t temp_storage_bytes = 0;
  auto& allocator = *c10::cuda::CUDACachingAllocator::get();
  auto num_nonzeros = allocator.allocate(sizeof(int) * num_chunks);
  for (int64_t idx = 0; idx < num_chunks; idx++) {
    int64_t remaining = std::min(chunk_size, self.numel() - idx * chunk_size);
    hipcub::TransformInputIterator<bool, NonZeroOp<scalar_t>, const scalar_t*> itr(
        self_.const_data_ptr<scalar_t>() + idx * chunk_size,
        NonZeroOp<scalar_t>());
    hipcub::DeviceReduce::Sum(
        nullptr,
        temp_storage_bytes,
        itr,
        ((int*)num_nonzeros.get()) + idx,
        remaining,
        stream);
    auto temp_storage = allocator.allocate(temp_storage_bytes);
    hipcub::DeviceReduce::Sum(
        temp_storage.get(),
        temp_storage_bytes,
        itr,
        ((int*)num_nonzeros.get()) + idx,
        remaining,
        stream);
  }
  auto pinned_num_nonzeros_h = at::detail::empty_cpu(
      {num_chunks}, /* size */
      c10::CppTypeToScalarType<int>(), /* dtype */
      std::nullopt, /* layout */
      std::nullopt, /* device */
      true, /* pin_memory */
      std::nullopt /* memory format */
  );
  at::cuda::memcpy_and_sync(
      (void*)pinned_num_nonzeros_h.const_data_ptr<int>(),
      num_nonzeros.get(),
      sizeof(int) * num_chunks,
      hipMemcpyDeviceToHost,
      stream);
  int64_t num_nonzeros_h = 0;

  for (int64_t idx = 0; idx < num_chunks; idx++) {
    num_nonzeros_h += (int)*(pinned_num_nonzeros_h.const_data_ptr<int>() + idx);
  }
  // num_nonzeros_h = (int)*(pinned_num_nonzeros_h.const_data_ptr<int>());
  // expected output size is num_nonzeros x ndim
  // we are producing output with size {num_nonzeros, ndim} and strides {1,
  // num_nonzeros} (that is, transposed ndim x num_nonzeros output) we are able
  // to directly use passed output with this size and strides, and we can also
  // (per contract) resize passed output with incorrect sizes anyway we want.
  // However, out with correct sizes and incorrect strides will have to be
  // copied to from the intermediate we've produced.
  bool need_to_copy = out.dim() == 2 && out.sizes()[0] == num_nonzeros_h &&
      out.sizes()[1] == self.dim() && !out.t().is_contiguous();
  at::Tensor out_temp = need_to_copy
      ? Tensor(
            at::detail::empty_cuda({self.dim(), num_nonzeros_h}, out.options()))
      : out.resize_({self.dim(), num_nonzeros_h});
  // Scalars are expected to produce output of size (1,0), so we can't write to
  // it
  int64_t curr_nonzeros = 0;
  if (self.dim() > 0) {
    for (int64_t idx = 0; idx < num_chunks; idx++) {
      int remaining = std::min(chunk_size, self.numel() - idx * chunk_size);

      hipcub::CountingInputIterator<int64_t> counting_itr(idx * chunk_size);
      hipcub::TransformInputIterator<bool, NonZeroOp<scalar_t>, const scalar_t*>
          itr(self_.const_data_ptr<scalar_t>() + idx * chunk_size,
              NonZeroOp<scalar_t>());
      temp_storage_bytes = 0;
      hipcub::DeviceSelect::Flagged(
          nullptr,
          temp_storage_bytes,
          counting_itr,
          itr,
          out_temp.mutable_data_ptr<int64_t>(),
          ((int*)num_nonzeros.get()) + idx,
          remaining,
          stream);
      auto temp_storage = allocator.allocate(temp_storage_bytes);
      hipcub::DeviceSelect::Flagged(
          temp_storage.get(),
          temp_storage_bytes,
          counting_itr,
          itr,
          out_temp.mutable_data_ptr<int64_t>() + curr_nonzeros,
          ((int*)num_nonzeros.get()) + idx,
          remaining,
          stream);
      curr_nonzeros +=
          (int)*(pinned_num_nonzeros_h.const_data_ptr<int>() + idx);
    }
    if (num_nonzeros_h > 0 && self.dim() > 1) {
      TensorDims<int64_t> dims;
      for (int i = 0; i < self.dim(); i++) {
        dims.sizes[i] = self.sizes()[i];
      }
      const int nthreads = 256;
      const int nblocks = (num_nonzeros_h + nthreads - 1) / nthreads;
      write_indices<<<nblocks, nthreads, 0, stream>>>(
          out_temp.mutable_data_ptr<int64_t>(),
          dims,
          self.dim(),
          num_nonzeros_h);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
  if (need_to_copy) {
    out.copy_(out_temp.t());
  } else {
    // transpose out so it is correct size
    Tensor out_ = out_temp.t();
    out.set_(out_);
  }
}

Tensor& nonzero_out_cuda(const Tensor& self, Tensor& out) {
  TORCH_CHECK(
      out.dtype() == at::kLong,
      "Expected object of scalar type ",
      at::kLong,
      " as out, but got ",
      out.dtype());
  TORCH_CHECK(
      self.device() == out.device(),
      "expected self and out to be on the same device, but got out on ",
      out.device(),
      " and self on ",
      self.device());
  TORCH_CHECK(
      self.dim() <= MAX_DIMS,
      "nonzero is not supported for tensor with more than ",
      MAX_DIMS,
      " dimensions");
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND4(
      at::ScalarType::ComplexHalf,
      at::ScalarType::Bool,
      at::ScalarType::BFloat16,
      at::ScalarType::Half,
      self.scalar_type(),
      "nonzero_cuda",
      [&] { nonzero_cuda_out_impl<scalar_t>(self, out); });
  return out;
}

Tensor nonzero_cuda(const Tensor& self) {
  Tensor out = at::detail::empty_cuda({0}, self.options().dtype(kLong));
  return at::native::nonzero_out_cuda(self, out);
}
} // namespace at::native
